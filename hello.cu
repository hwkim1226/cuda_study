#include <iostream>
#include <stdio.h>
#include <fstream>
#include <cmath>

#include "hip/hip_runtime.h"


using namespace std;

void __global__ sampleAdd(int *src, int* dst);


int main()
{
    int *h_arr;
    int *d_arr;
    int *d_add;
    int *h_add;

    hipHostMalloc((void**)&h_arr, 128*sizeof(int), hipHostMallocDefault);
    hipMalloc((void**)&d_arr, 128*sizeof(int));
    hipMalloc((void**)&d_add, 128*sizeof(int));
    hipHostMalloc((void**)&h_add, 128*sizeof(int), hipHostMallocDefault);


    for (int i = 0; i < 128; i++)
    {
        h_arr[i] = rand() % 128;
    }

    dim3 blocks(128, 1, 1);
    dim3 grids(1, 1, 1);
    hipMemcpy(d_arr, h_arr, 128*sizeof(int), hipMemcpyHostToDevice);
    sampleAdd<<<grids, blocks>>>(d_arr, d_add);
    hipMemcpy(h_add, d_add, 128*sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 128; i++)
    {
        cout << i << "th input is " << h_arr[i] << ", and output is " << h_add[i] << endl;
    }

    
    return 0;
}


void __global__ sampleAdd(int *src, int* dst)
{
    int a = threadIdx.x;
    dst[a] += src[a] + a;
}